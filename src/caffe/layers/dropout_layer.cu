#include "hip/hip_runtime.h"
#include <algorithm>
#include <limits>
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {


template <typename Dtype, typename Mtype>
__global__ void DropoutForward(const int n, const Dtype* in,
    const unsigned int* mask, const unsigned int threshold, const Mtype scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = Get<Dtype>( Get<Mtype>(in[index]) * (mask[index] > threshold) * scale );
  }
}

template <typename Dtype, typename Mtype>
void DropoutLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (this->phase_ == TRAIN) {
    unsigned int* mask =
        static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
    caffe_gpu_rng_uniform(count, mask);
    CUDA_POST_KERNEL_CHECK;
    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
    DropoutForward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, uint_thres_, scale_, top_data);
  } else {
    caffe_copy<Dtype,Mtype>(count, bottom_data, top_data);
  }
}

template <typename Dtype, typename Mtype>
__global__ void DropoutBackward(const int n, const Dtype* in_diff,
    const unsigned int* mask, const unsigned int threshold, const Mtype scale,
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = Get<Dtype>( Get<Mtype>(in_diff[index]) * scale * (mask[index] > threshold) );
  }
}

template <typename Dtype, typename Mtype>
void DropoutLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype,Mtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (this->phase_ == TRAIN) {
      const unsigned int* mask =
          static_cast<const unsigned int*>(rand_vec_.gpu_data());
      const int count = bottom[0]->count();
      // NOLINT_NEXT_LINE(whitespace/operators)
      DropoutBackward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
          count, top_diff, mask, uint_thres_, scale_, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy<Dtype,Mtype>(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DropoutLayer);


}  // namespace caffe
