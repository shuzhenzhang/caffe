#include "hip/hip_runtime.h"
// TanH neuron activation function layer.
// Adapted from ReLU layer code written by Yangqing Jia

#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
__global__ void TanHForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = Get<Dtype>( tanh(Get<Mtype>(in[index])) );
  }
}

template <typename Dtype, typename Mtype>
void TanHLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  TanHForward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype, typename Mtype>
__global__ void TanHBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    Mtype tanhx = Get<Mtype>(out_data[index]);
    out_diff[index] = Get<Dtype>( Get<Mtype>(in_diff[index]) * (1 - tanhx * tanhx) );
  }
}

template <typename Dtype, typename Mtype>
void TanHLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype,Mtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    TanHBackward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TanHLayer);


}  // namespace caffe
