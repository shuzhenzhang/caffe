
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/gpu_memory.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

// Those defines serve single purpose to keep sane C++ formatting
// in presence of <80 characters rule
#define cudnnConvFwd                       cudnnConvolutionForward
#define cudnnConvBwdBias                   cudnnConvolutionBackwardBias
#define cudnnConvBwdFilter                 cudnnConvolutionBackwardFilter
#define cudnnConvBwdData                   cudnnConvolutionBackwardData

namespace caffe {

  __global__ void sync_conv_groups() { }

template <typename Dtype, typename Mtype>
void CuDNNConvolutionLayer<Dtype,Mtype>::Forward_gpu(
    const vector<Blob<Dtype,Mtype>*>& bottom, const vector<Blob<Dtype,Mtype>*>& top) {
    const Dtype* weight = this->blobs_[0]->gpu_data();
    for (int i = 0; i < bottom.size(); ++i) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* top_data = top[i]->mutable_gpu_data();


      // Forward through cuDNN in parallel over groups.
      for (int g = 0; g < this->group_; g++) {
        gpu_memory::allocate(&workspaceData, workspace_fwd_sizes_[i]);
        // Filters.
        CUDNN_CHECK(cudnnConvFwd(Caffe::cudnn_handle(),
                                 cudnn::dataType<Dtype>::one,
                                 bottom_descs_[i],
                                 bottom_data + bottom_offset_ * g,
                                 fwd_filter_desc_,
                                 weight + this->weight_offset_ * g,
                                 fwd_conv_descs_[i],
                                 fwd_algo_[i], workspaceData,
                                 workspace_fwd_sizes_[i],
                                 cudnn::dataType<Dtype>::zero,
                                 top_descs_[i],
                                 top_data + top_offset_ * g));
        int array_length;
        int padA[10];
        int strideA[10];
        int upscaleA[10];
        cudnnConvolutionMode_t mode;
        cudnnDataType_t dataType;
        CUDNN_CHECK(cudnnGetConvolutionNdDescriptor(fwd_conv_descs_[i],1,&array_length,
                                                    padA, strideA, upscaleA, &mode, &dataType));

        gpu_memory::deallocate(workspaceData);
        workspaceData = NULL;
        // Bias.
        if (this->bias_term_) {
          const Dtype* bias_data = this->blobs_[1]->gpu_data();
          CUDNN_CHECK(cudnnAddTensor(Caffe::cudnn_handle(),
                                        cudnn::dataType<Dtype>::one,
                                        bias_desc_,
                                        bias_data + bias_offset_ * g,
                                        cudnn::dataType<Dtype>::one,
                                        top_descs_[i],
                                        top_data + top_offset_ * g));
        }

      }

      // Synchronize the work across groups, each of which went into its own
      // stream, by launching an empty kernel into the default (null) stream.
      // NOLINT_NEXT_LINE(whitespace/operators)
      if(this->group_ > 1) {
          CUDA_CHECK(cudaStreamSynchronize(cudaStreamLegacy));
      }
    }
  }


template <typename Dtype, typename Mtype>
void CuDNNConvolutionLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype,Mtype>*>& bottom) {
    const Dtype* weight = NULL;
    Dtype* weight_diff = NULL;

    if (this->param_propagate_down_[0]) {
      weight = this->blobs_[0]->gpu_data();
      weight_diff = this->blobs_[0]->mutable_gpu_diff();
    caffe_gpu_set<Dtype,Mtype>(this->blobs_[0]->count(), Mtype(0), weight_diff);
    }
    Dtype* bias_diff = NULL;

    if (this->bias_term_ && this->param_propagate_down_[1]) {
      bias_diff = this->blobs_[1]->mutable_gpu_diff();
    caffe_gpu_set<Dtype,Mtype>(this->blobs_[1]->count(), Mtype(0), bias_diff);
    }

    for (int i = 0; i < top.size(); ++i) {
      const Dtype* top_diff = top[i]->gpu_diff();

      // Backward through cuDNN in parallel over groups and gradients.
      for (int g = 0; g < this->group_; g++) {
        // Gradient w.r.t. bias.
        if (this->bias_term_ && this->param_propagate_down_[1]) {
          CUDNN_CHECK(cudnnConvBwdBias(Caffe::cudnn_handle(),
                                       cudnn::dataType<Dtype>::one,
                                       top_descs_[i],
                                       top_diff + top_offset_ * g,
                                       cudnn::dataType<Dtype>::one,
                                       bias_desc_,
                                       bias_diff + bias_offset_ * g));
        }

        // Gradient w.r.t. weights.
        if (this->param_propagate_down_[0]) {
          gpu_memory::allocate(&workspaceData,
                               workspace_bwd_filter_sizes_[i]);
          const Dtype* bottom_data = bottom[i]->gpu_data();
          CUDNN_CHECK(cudnnConvBwdFilter(Caffe::cudnn_handle(),
                                         cudnn::dataType<Dtype>::one,
                                         bottom_descs_[i],
                                         bottom_data + bottom_offset_ * g,
                                         top_descs_[i],
                                         top_diff + top_offset_ * g,
                                         bwd_conv_descs_[i],
                                         bwd_filter_algo_[i],
                                         workspaceData,
                                         workspace_bwd_filter_sizes_[i],
                                         cudnn::dataType<Dtype>::one,
                                         bwd_filter_desc_,
                                         weight_diff + weight_offset_ * g));
          gpu_memory::deallocate(workspaceData);
          workspaceData = NULL;
        }

        // Gradient w.r.t. bottom data.
        if (propagate_down[i]) {
          if (weight == NULL) {
            weight = this->blobs_[0]->gpu_data();
          }
          Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
          gpu_memory::allocate(&workspaceData,
                               workspace_bwd_data_sizes_[i]);
          CUDNN_CHECK(cudnnConvBwdData(Caffe::cudnn_handle(),
                                       cudnn::dataType<Dtype>::one,
                                       bwd_filter_desc_,
                                       weight + this->weight_offset_ * g,
                                       top_descs_[i],
                                       top_diff + top_offset_ * g,
                                       bwd_conv_descs_[i],
                                       bwd_data_algo_[i], workspaceData,
                                       workspace_bwd_data_sizes_[i],
                                       cudnn::dataType<Dtype>::zero,
                                       bottom_descs_[i],
                                       bottom_diff + bottom_offset_ * g));
          gpu_memory::deallocate(workspaceData);
          workspaceData = NULL;
        }
      }

      // Synchronize the work across groups, each of which went into its own
      // stream, by launching an empty kernel into the default (null) stream.
      // NOLINT_NEXT_LINE(whitespace/operators)
      if(this->group_ > 1) {
          CUDA_CHECK(cudaStreamSynchronize(cudaStreamLegacy));
      }
    }
  }

  INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
