#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
__global__ void ReLUForward(const int n, const Dtype* in, Dtype* out,
    Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : Dtype(in[index] * negative_slope);
  }
}

template <typename Dtype, typename Mtype>
void ReLULayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype negative_slope(this->layer_param_.relu_param().negative_slope());
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLUForward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, negative_slope);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype, typename Mtype>
__global__ void ReLUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Mtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = Get<Dtype>( Get<Mtype>(in_diff[index]) * ((Get<Mtype>(in_data[index]) > 0)
        + (Get<Mtype>(in_data[index]) <= 0) * negative_slope) );
  }
}

template <typename Dtype, typename Mtype>
void ReLULayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype,Mtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Mtype negative_slope(this->layer_param_.relu_param().negative_slope());
    // NOLINT_NEXT_LINE(whitespace/operators)
    ReLUBackward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, negative_slope);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ReLULayer);


}  // namespace caffe
