#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
__global__ void MaxForward(const int nthreads, const Dtype* bottom_data_a,
    const Dtype* bottom_data_b, const int blob_idx, Dtype* top_data,
    int* mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Mtype maxval(- maxDtype<Dtype>());
    int maxidx = -1;
    if (Get<Mtype>(bottom_data_a[index]) > Get<Mtype>(bottom_data_b[index])) {
      // only update for very first bottom_data blob (blob_idx == 0)
      if (blob_idx == 0) {
        maxval = Get<Mtype>(bottom_data_a[index]);
        top_data[index] = Get<Dtype>(maxval);
        maxidx = blob_idx;
        mask[index] = maxidx;
      }
    } else {
      maxval = Get<Mtype>(bottom_data_b[index]);
      top_data[index] = Get<Dtype>(maxval);
      maxidx = blob_idx + 1;
      mask[index] = maxidx;
    }
  }
}

template <typename Dtype, typename Mtype>
void EltwiseLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  int* mask = NULL;
  const int count = top[0]->count();
  Dtype* top_data = top[0]->mutable_gpu_data();
  switch (op_) {
  case EltwiseParameter_EltwiseOp_PROD:
    caffe_gpu_mul<Dtype,Mtype>(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
        top_data);
    for (int i = 2; i < bottom.size(); ++i) {
      caffe_gpu_mul<Dtype,Mtype>(count, top_data, bottom[i]->gpu_data(), top_data);
    }
    break;
  case EltwiseParameter_EltwiseOp_SUM:
    caffe_gpu_set<Dtype,Mtype>(count, Mtype(0.), top_data);
    // TODO(shelhamer) does cuBLAS optimize to sum for coeff = 1?
    for (int i = 0; i < bottom.size(); ++i) {
      caffe_gpu_axpy<Dtype,Mtype>(count, Get<Mtype>(coeffs_[i]), bottom[i]->gpu_data(), top_data);
    }
    break;
  case EltwiseParameter_EltwiseOp_MAX:
    mask = max_idx_.mutable_gpu_data();
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxForward<Dtype,Mtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom[0]->gpu_data(), bottom[1]->gpu_data(), 0, top_data, mask);
    for (int i = 2; i < bottom.size(); ++i) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      MaxForward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, top_data, bottom[i]->gpu_data(), i-1, top_data, mask);
    }
    break;
  default:
    LOG(FATAL) << "Unknown elementwise operation.";
  }
}

template <typename Dtype, typename Mtype>
__global__ void MaxBackward(const int nthreads, const Dtype* top_diff,
    const int blob_idx, const int* mask, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Mtype gradient(0.);
    if (mask[index] == blob_idx) {
      gradient += Get<Mtype>(top_diff[index]);
    }
    bottom_diff[index] = Get<Dtype>(gradient);
  }
}

template <typename Dtype, typename Mtype>
void EltwiseLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype,Mtype>*>& bottom) {
  const int* mask = NULL;
  const int count = top[0]->count();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  for (int i = 0; i < bottom.size(); ++i) {
    if (propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      switch (op_) {
      case EltwiseParameter_EltwiseOp_PROD:
        if (stable_prod_grad_) {
          bool initialized = false;
          for (int j = 0; j < bottom.size(); ++j) {
            if (i == j) { continue; }
            if (!initialized) {
              caffe_copy<Dtype,Mtype>(count, bottom[j]->gpu_data(), bottom_diff);
              initialized = true;
            } else {
              caffe_gpu_mul<Dtype,Mtype>(count, bottom[j]->gpu_data(), bottom_diff,
                            bottom_diff);
            }
          }
        } else {
          caffe_gpu_div<Dtype,Mtype>(count, top_data, bottom_data, bottom_diff);
        }
        caffe_gpu_mul<Dtype,Mtype>(count, bottom_diff, top_diff, bottom_diff);
        break;
      case EltwiseParameter_EltwiseOp_SUM:
        if (Get<Mtype>(coeffs_[i]) == Mtype(1.)) {
          caffe_copy<Dtype,Mtype>(count, top_diff, bottom_diff);
        } else {
          caffe_gpu_scale<Dtype,Mtype>(count, Get<Mtype>(coeffs_[i]), top_diff, bottom_diff);
        }
        break;
      case EltwiseParameter_EltwiseOp_MAX:
        mask = max_idx_.gpu_data();
        MaxBackward<Dtype,Mtype>  // NOLINT_NEXT_LINE(whitespace/operators)
            <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, top_diff, i, mask, bottom_diff);
        break;
      default:
        LOG(FATAL) << "Unknown elementwise operation.";
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EltwiseLayer);

}  // namespace caffe
