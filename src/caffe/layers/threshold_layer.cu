#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
__global__ void ThresholdForward(const int n, const Mtype threshold,
    const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = Get<Dtype>( Get<Mtype>(in[index]) > threshold ? 1 : 0 );
  }
}

template <typename Dtype, typename Mtype>
void ThresholdLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ThresholdForward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, threshold_, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FORWARD(ThresholdLayer);
INSTANTIATE_LAYER_GPU_FORWARD_FF(ThresholdLayer);


}  // namespace caffe
