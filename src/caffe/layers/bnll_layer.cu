#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
__global__ void BNLLForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    Mtype in_index = Get<Mtype>(in[index]);
    out[index] = Get<Dtype>( in_index > 0 ?
        in_index + log(1. + exp(-in_index)) :
        log(1. + exp(in_index)) );
  }
}

template <typename Dtype, typename Mtype>
void BNLLLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  BNLLForward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype, typename Mtype>
__global__ void BNLLBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    Mtype expval(exp(min(Get<Mtype>(in_data[index]), Mtype( 50. ))));
    out_diff[index] = Get<Dtype>( Get<Mtype>(in_diff[index]) * expval / (expval + 1.) );
  }
}

template <typename Dtype, typename Mtype>
void BNLLLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype,Mtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    BNLLBackward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(BNLLLayer);


}  // namespace caffe
