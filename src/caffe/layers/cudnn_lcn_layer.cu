
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/gpu_memory.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
void CuDNNLCNLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  gpu_memory::allocate(&this->tempData1, this->tempDataSize);
  gpu_memory::allocate(&this->tempData2, this->tempDataSize);

  CUDNN_CHECK(cudnnDivisiveNormalizationForward(
        Caffe::cudnn_handle(), norm_desc_, CUDNN_DIVNORM_PRECOMPUTED_MEANS,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        NULL,  // srcMeansData
        this->tempData1, this->tempData2,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data) );

  gpu_memory::deallocate(this->tempData1);
  gpu_memory::deallocate(this->tempData2);
}

template <typename Dtype, typename Mtype>
void CuDNNLCNLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype,Mtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  gpu_memory::allocate(&this->tempData1, this->tempDataSize);
  gpu_memory::allocate(&this->tempData2, this->tempDataSize);

  CUDNN_CHECK(cudnnDivisiveNormalizationBackward(
        Caffe::cudnn_handle(), norm_desc_, CUDNN_DIVNORM_PRECOMPUTED_MEANS,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        NULL, top_diff,  // NULL - srcMeansData
        this->tempData1, this->tempData2,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff,
        NULL) );

  gpu_memory::deallocate(this->tempData1);
  gpu_memory::deallocate(this->tempData2);
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNLCNLayer);

}  // namespace caffe
#endif
