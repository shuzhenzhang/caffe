#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
void PowerLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // Special case where we can ignore the input: scale or power is 0.
  if (diff_scale_ == Mtype(0.)) {
    Mtype value = Mtype(power_ == 0 ? 1.0 : pow(shift_, power_));
    caffe_gpu_set<Dtype,Mtype>(count, value, top_data);
    return;
  }
  const Dtype* bottom_data = bottom[0]->gpu_data();
  caffe_copy<Dtype,Mtype>(count, bottom_data, top_data);
  if (scale_ != Mtype(1)) {
    caffe_gpu_scal<Dtype,Mtype>(count, scale_, top_data);
  }
  if (shift_ != Mtype(0)) {
    caffe_gpu_add_scalar<Dtype,Mtype>(count, shift_, top_data);
  }
  if (power_ != Mtype(1)) {
    caffe_gpu_powx<Dtype,Mtype>(count, top_data, power_, top_data);
  }
}

template <typename Dtype, typename Mtype>
void PowerLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype,Mtype>*>& bottom) {
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    const Dtype* top_diff = top[0]->gpu_diff();
    if (diff_scale_ == Mtype(0) || power_ == Mtype(1)) {
      caffe_gpu_set<Dtype,Mtype>(count, diff_scale_, bottom_diff);
    } else {
      const Dtype* bottom_data = bottom[0]->gpu_data();
      // Compute dy/dx = scale * power * (shift + scale * x)^(power - 1)
      //               = diff_scale * y / (shift + scale * x)
      if (power_ == Mtype(2)) {
        // Special case for y = (shift + scale * x)^2
        //     -> dy/dx = 2 * scale * (shift + scale * x)
        //              = diff_scale * shift + diff_scale * scale * x
        caffe_gpu_axpby<Dtype,Mtype>(count, diff_scale_ * scale_, bottom_data,
            Mtype(0), bottom_diff);
        if (shift_ != Mtype(0)) {
          caffe_gpu_add_scalar<Dtype,Mtype>(count, diff_scale_ * shift_, bottom_diff);
        }
      } else if (shift_ == Mtype(0)) {
        // Special case for y = (scale * x)^power
        //     -> dy/dx = scale * power * (scale * x)^(power - 1)
        //              = scale * power * (scale * x)^power * (scale * x)^(-1)
        //              = power * y / x
        const Dtype* top_data = top[0]->gpu_data();
        caffe_gpu_div<Dtype,Mtype>(count, top_data, bottom_data, bottom_diff);
        caffe_gpu_scal<Dtype,Mtype>(count, power_, bottom_diff);
      } else {
        caffe_copy<Dtype,Mtype>(count, bottom_data, bottom_diff);
        if (scale_ != Mtype(1)) {
          caffe_gpu_scal<Dtype,Mtype>(count, scale_, bottom_diff);
        }
        if (shift_ != Mtype(0)) {
          caffe_gpu_add_scalar<Dtype,Mtype>(count, shift_, bottom_diff);
        }
        const Dtype* top_data = top[0]->gpu_data();
        caffe_gpu_div<Dtype,Mtype>(count, top_data, bottom_diff, bottom_diff);
        if (diff_scale_ != Mtype(1)) {
          caffe_gpu_scal<Dtype,Mtype>(count, diff_scale_, bottom_diff);
        }
      }
    }
    caffe_gpu_mul<Dtype,Mtype>(count, top_diff, bottom_diff, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PowerLayer);


}  // namespace caffe
