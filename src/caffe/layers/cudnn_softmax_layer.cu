
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
void CuDNNSoftmaxLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
    SoftmaxLayer<Dtype,Mtype>::Forward_gpu(bottom, top);
}

template <typename Dtype, typename Mtype>
void CuDNNSoftmaxLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype,Mtype>*>& bottom) {
    SoftmaxLayer<Dtype,Mtype>::Backward_gpu(top, propagate_down, bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNSoftmaxLayer);

}  // namespace caffe
#endif
