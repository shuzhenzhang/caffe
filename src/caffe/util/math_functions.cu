#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <>
void caffe_gpu_gemm<float,float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double,double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<float16,float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float16* A, const float16* B, const float beta,
    float16* C) {
  // Note that cublas follows fortran order.
  const int lda = (TransA == CblasNoTrans) ? K : M;
  const int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(cublasSgemmEx(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, CAFFE_DATA_HALF, ldb, A, CAFFE_DATA_HALF,
      lda, &beta, C, CAFFE_DATA_HALF, N));
}

template <>
void caffe_gpu_gemm<float16,float16>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float16 alpha, const float16* A, const float16* B, const float16 beta,
    float16* C) {
  // Note that cublas follows fortran order.
  const int lda = (TransA == CblasNoTrans) ? K : M;
  const int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasHgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha.data, &B->data, ldb, &A->data,
      lda, &beta.data, &C->data, N));
}

template <>
void caffe_gpu_gemv<float,float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double,double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

#if !NATIVE_FP16
    template <>
void caffe_gpu_gemv<float16, float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float16* A, const float16* x,
    const float beta, float16* y) {
    hipblasOperation_t cuTransA =
        (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    int m = (cuTransA == HIPBLAS_OP_N) ? N : M;
    int k = (cuTransA == HIPBLAS_OP_N) ? M : N;
    int LDA = (cuTransA == HIPBLAS_OP_N) ? m : k;
//    int LDB = (cuTransA == HIPBLAS_OP_N) ? k : m;
    int LDC = m;
    
    CUBLAS_CHECK(cublasSgemmEx(Caffe::cublas_handle(), cuTransA, HIPBLAS_OP_N,
                               m, 1, k, &alpha, A, CAFFE_DATA_HALF, LDA, x, CAFFE_DATA_HALF, k, &beta,
                               y, CAFFE_DATA_HALF, LDC));
}
#else

template <>
void caffe_gpu_gemv<float16, float16>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float16 alpha, const float16* A, const float16* x,
    const float16 beta, float16* y) {

    float alpha_fp32 = cpu_half2float(alpha);
    float beta_fp32 = cpu_half2float(beta);
    hipblasOperation_t cuTransA =
        (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    int m = (cuTransA == HIPBLAS_OP_N) ? N : M;
    int k = (cuTransA == HIPBLAS_OP_N) ? M : N;
    int LDA = (cuTransA == HIPBLAS_OP_N) ? m : k;
//    int LDB = (cuTransA == HIPBLAS_OP_N) ? k : m;
    int LDC = m;
    
    CUBLAS_CHECK(cublasSgemmEx(Caffe::cublas_handle(), cuTransA, HIPBLAS_OP_N,
                               m, 1, k, &alpha_fp32, A, CAFFE_DATA_HALF, LDA, x, CAFFE_DATA_HALF, k, &beta_fp32,
                               y, CAFFE_DATA_HALF, LDC));
}
#endif

template <>
void caffe_gpu_axpy<float,float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double,double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <typename T_STORE, typename T_MATH>
__global__
void axpy_kernel(const int N, const T_MATH alpha, const T_STORE *x, T_STORE *y)
{
  for (int idx = threadIdx.x + blockDim.x*blockIdx.x; idx < N; idx += blockDim.x*gridDim.x) {
    y[idx] = Get<T_STORE>( alpha * Get<T_MATH>(x[idx]) + Get<T_MATH>(y[idx]) );
  }
}

template <>
void caffe_gpu_axpy<float16,float>(const int N, const float alpha, const float16* x, float16 *y) {
  axpy_kernel<float16,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, alpha, x, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_axpy<float16,float16>(const int N, const float16 alpha, const float16* x, float16 *y) {
  axpy_kernel<float16,float16><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, alpha, x, y);
  CUDA_POST_KERNEL_CHECK;
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float,float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double,double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <typename T_STORE, typename T_MATH>
__global__
void scal_kernel(const int N, const T_MATH alpha, T_STORE *X)
{
  for (int idx = threadIdx.x + blockDim.x*blockIdx.x; idx < N; idx += blockDim.x*gridDim.x) {
    X[idx] = Get<T_STORE>( alpha * Get<T_MATH>(X[idx]));
  }
}

template <>
void caffe_gpu_scal<float16,float>(const int N, const float alpha, float16 *X) {
  scal_kernel<float16,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, alpha, X);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_scal<float16,float16>(const int N, const float16 alpha, float16 *X) {
  scal_kernel<float16,float16><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, alpha, X);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_axpby<float,float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float,float>(N, beta, Y);
  caffe_gpu_axpy<float,float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double,double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double,double>(N, beta, Y);
  caffe_gpu_axpy<double,double>(N, alpha, X, Y);
}

template <typename T_STORE, typename T_MATH>
__global__
void axpby_kernel(const int N, const T_MATH alpha, const T_STORE* X,
    const T_MATH beta, T_STORE* Y)
{
  CUDA_KERNEL_LOOP(idx, N) {
    Y[idx] = Get<T_STORE>( alpha * Get<T_MATH>(X[idx]) + beta * Get<T_MATH>(Y[idx]) );
  }
}

template <>
void caffe_gpu_axpby<float16,float>(const int N, const float alpha, const float16* X,
    const float beta, float16* Y)
{
  axpby_kernel<float16,float><<<CAFFE_GET_BLOCKS(N),CAFFE_CUDA_NUM_THREADS>>>(N,alpha,X,beta,Y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_axpby<float16,float16>(const int N, const float16 alpha, const float16* X,
    const float16 beta, float16* Y)
{
  axpby_kernel<float16,float><<<CAFFE_GET_BLOCKS(N),CAFFE_CUDA_NUM_THREADS>>>(N,alpha,X,beta,Y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_dot<float,float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double,double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

struct float16_dot_reduce {
  __host__ __device__
  float operator()(const float& x, const float& y)
  {
    return x + y;
  }
};

struct float16_dot_mult {
  __host__ __device__
  float operator()(float16& x, float16& y)
  {
    return Get<float>(x) * Get<float>(y);
  }
};

template <typename Dtype, typename Mtype>
__global__
void gpu_dot_kernel(const int N, const Dtype *x, const Dtype *y, Mtype *out)
{
  __shared__ Mtype cache[256];

  const int tidx = threadIdx.x;
  cache[tidx] = Get<Mtype>(0);
  for (int i=tidx; i<N; i+=blockDim.x) {
    cache[tidx] += Get<Mtype>(x[i]) * Get<Mtype>(y[i]);
  }
  __syncthreads();
  for (int s=128; s > 0; s >>= 1) {
    if (tidx < s) cache[tidx] += cache[tidx+s];
    __syncthreads();
  }

  if (tidx == 0) *out = cache[tidx];
}

template <>
void caffe_gpu_dot<float16, float>(const int n, const float16* x, const float16* y,
    float *out)
{
  // float ret = thrust::inner_product(x, x+n, y, init, float16_dot_reduce(), float16_dot_mult());
  // *out = ret;

  float *res;
  hipMalloc(&res, sizeof(float));
  gpu_dot_kernel<float16,float><<<1,256>>>(n, x, y, res);
  CUDA_POST_KERNEL_CHECK;
  hipMemcpy(out,res,sizeof(float), hipMemcpyDeviceToHost);
  hipFree(res);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_dot<float16, float16>(const int n, const float16* x, const float16* y, float16 *out)
{
  float16 *res;
  hipMalloc(&res, sizeof(float16));
  gpu_dot_kernel<float16,float16><<<1,256>>>(n, x, y, res);
  CUDA_POST_KERNEL_CHECK;
  hipMemcpy(out, res, sizeof(float16), hipMemcpyDeviceToHost);
  hipFree(res);
  CUDA_POST_KERNEL_CHECK;
}


template <>
void caffe_gpu_asum<float,float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double,double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

struct float16_asum_reduce
{
  __host__ __device__
  float operator()(const float& a, const float16& b)
  {
    return a + fabs(Get<float>(b));
  }
};

template <typename Dtype, typename Mtype>
__global__
void gpu_asum_kernel(const int N, const Dtype *x, Mtype *out)
{
  __shared__ Mtype cache[256];

  const int tidx = threadIdx.x;
  cache[tidx] = Get<Mtype>(0);
  for (int i=tidx; i<N; i+=blockDim.x) {
    cache[tidx] += Get<Mtype>(fabs(x[i]));
  }
  __syncthreads();
  for (int s=128; s > 0; s >>= 1) {
    if (tidx < s) cache[tidx] += cache[tidx+s];
    __syncthreads();
  }

  if (tidx == 0) *out = cache[tidx];
}

template <>
void caffe_gpu_asum<float16,float>(const int n, const float16* x, float* y)
{
  // float init = 0.0f;
  // float result = thrust::reduce(x, x+n, init, float16_asum_reduce());
  // *y = result;
  float *res;
  hipMalloc(&res, sizeof(float));
  gpu_asum_kernel<float16,float><<<1,256>>>(n,x,res);
  CUDA_POST_KERNEL_CHECK;
  hipMemcpy(y,res,sizeof(float),hipMemcpyDeviceToHost);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_asum<float16,float16>(const int n, const float16* x, float16* y)
{
  float16 *res;
  hipMalloc(&res, sizeof(float16));
  gpu_asum_kernel<float16,float16><<<1,256>>>(n,x,res);
  CUDA_POST_KERNEL_CHECK;
  hipMemcpy(y,res,sizeof(float16),hipMemcpyDeviceToHost);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_scale<float,float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double,double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename T_STORE, typename T_MATH>
__global__
void scale_kernel(const int n, const T_MATH alpha, const T_STORE* x, T_STORE* y)
{
  CUDA_KERNEL_LOOP(idx, n) {
    y[idx] = Get<T_STORE>( alpha * Get<T_MATH>(x[idx]) );
  }
}

template <>
void caffe_gpu_scale<float16,float>(const int n, const float alpha, const float16 *x,
    float16 *y)
{
  scale_kernel<float16,float><<<CAFFE_GET_BLOCKS(n),CAFFE_CUDA_NUM_THREADS>>>(n,alpha,x,y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_scale<float16,float16>(const int n, const float16 alpha, const float16 *x,
                            float16* y) {
  scale_kernel<float16,float16><<<CAFFE_GET_BLOCKS(n),CAFFE_CUDA_NUM_THREADS>>>(n,alpha,x,y);
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype, typename Mtype>
__global__ void set_kernel(const int n, const Mtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>(alpha);
  }
}

template <typename Dtype, typename Mtype>
void caffe_gpu_set(const int N, const Mtype alpha, Dtype* Y) {
  if (alpha == 0.) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype,Mtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
  CUDA_POST_KERNEL_CHECK;
}

template void caffe_gpu_set<int,int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float,float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double,double>(const int N, const double alpha, double* Y);
template void caffe_gpu_set<float16,CAFFE_FP16_MTYPE>(const int N,
    const CAFFE_FP16_MTYPE alpha, float16* Y);

template <typename Dtype, typename Mtype>
__global__ void add_scalar_kernel(const int n, const Mtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>(alpha + Get<Mtype>(y[index]));
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_add_scalar(const int N, const CAFFE_FP16_MTYPE alpha, float16* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float16,CAFFE_FP16_MTYPE><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype, typename Mtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( Get<Mtype>(a[index]) + Get<Mtype>(b[index]) );
  }
}

template <>
void caffe_gpu_add<float,float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_add<double,double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_add<float16,CAFFE_FP16_MTYPE>(const int N, const float16* a, const float16* b,
    float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float16,CAFFE_FP16_MTYPE><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype, typename Mtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( Get<Mtype>(a[index]) - Get<Mtype>(b[index]) );
  }
}

template <>
void caffe_gpu_sub<float,float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_sub<double,double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_sub<float16,CAFFE_FP16_MTYPE>(const int N, const float16* a, const float16* b,
    float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float16,CAFFE_FP16_MTYPE><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype, typename Mtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( Get<Mtype>(a[index]) * Get<Mtype>(b[index]) );
  }
}

template <>
void caffe_gpu_mul<float, float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_mul<double,double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_mul<float16,float>(const int N, const float16* a,
    const float16* b, float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float16,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_mul<float16,float16>(const int N, const float16* a,
    const float16* b, float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float16,float16><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype, typename Mtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( Get<Mtype>(a[index]) / Get<Mtype>(b[index]) );
  }
}

template <>
void caffe_gpu_div<float,float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_div<double,double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_div<float16,float>(const int N, const float16* a,
    const float16* b, float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float16,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}


template <>
void caffe_gpu_div<float16,float16>(const int N, const float16* a,
    const float16* b, float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float16,float16><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype, typename Mtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( abs(Get<Mtype>(a[index])) );
  }
}

template <>
void caffe_gpu_abs<float,float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_abs<double,double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_abs<float16,float>(const int N, const float16* a, float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float16,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
  CUDA_POST_KERNEL_CHECK;
}


template <>
void caffe_gpu_abs<float16,float16>(const int N, const float16* a, float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float16,float16><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype, typename Mtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( exp(Get<Mtype>(a[index])) );
  }
}

template <>
void caffe_gpu_exp<float,float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_exp<double,double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_exp<float16,float16>(const int N, const float16* a, float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float16,float16><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_exp<float16,float>(const int N, const float16* a, float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float16,float16><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<float16>(const int N, const float16* a, float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float16><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype, typename Mtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Mtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = Get<Dtype>( pow(Get<Mtype>(a[index]), alpha) );
  }
}

template <>
void caffe_gpu_powx<float,float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_powx<double,double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double,double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_powx<float16,float16>(const int N, const float16* a,
    const float16 alpha, float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float16,float16><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
  CUDA_POST_KERNEL_CHECK;
}


template <>
void caffe_gpu_powx<float16,float>(const int N, const float16* a,
    const float alpha, float16* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float16,float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
  CUDA_POST_KERNEL_CHECK;
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = Get<Dtype>( (Mtype(0) < Get<Mtype>(x[index]))
                                      - (Get<Mtype>(x[index]) < Mtype(0))) );
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = Get<Dtype>( signbit(Get<Mtype>(x[index]))) );

__global__ void popc_kernel(const int n, const float* a,
    const float* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popc(static_cast<uint32_t>(a[index]) ^
                      static_cast<uint32_t>(b[index]));
  }
}

__global__ void popcll_kernel(const int n, const double* a,
    const double* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popcll(static_cast<uint64_t>(a[index]) ^
                      static_cast<uint64_t>(b[index]));
  }
}

__global__ void popch_kernel(const int n, const float16* a,
    const float16* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popc(static_cast<uint32_t>(Get<float>(a[index])) ^
                      static_cast<uint32_t>(Get<float>(b[index])));
  }
}

template <>
uint32_t caffe_gpu_hamming_distance<float>(const int n, const float* x,
                                  const float* y) {
  // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
  // TestHammingDistanceGPU in test_math_functions.cpp).
  NOT_IMPLEMENTED;
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popc_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  return thrust::reduce(popcounts.begin(), popcounts.end(),
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

template <>
uint32_t caffe_gpu_hamming_distance<double>(const int n, const double* x,
                                   const double* y) {
  // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
  // TestHammingDistanceGPU in test_math_functions.cpp).
  NOT_IMPLEMENTED;
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popcll_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  return thrust::reduce(popcounts.begin(), popcounts.end(),
                        /* NOLINT_NEXT_LINE(build/include_what_you_use) */
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

template <>
uint32_t caffe_gpu_hamming_distance<float16>(const int n, const float16* x,
                                   const float16* y) {
  // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
  // TestHammingDistanceGPU in test_math_functions.cpp).
  NOT_IMPLEMENTED;
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popch_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  return thrust::reduce(popcounts.begin(), popcounts.end(),
                        /* NOLINT_NEXT_LINE(build/include_what_you_use) */
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

template <typename T_IN, typename T_OUT>
__global__
void convert_kernel(const int n, const T_IN* in, T_OUT* out)
{
  for (int idx=threadIdx.x+blockIdx.x*blockDim.x; idx<n; idx+=blockDim.x*gridDim.x) {
    out[idx] = Get<T_OUT>(in[idx]);
  }
}

template <typename T_IN, typename T_OUT>
void caffe_gpu_convert(const int n, const T_IN* in, T_OUT* out)
{
  convert_kernel<T_IN,T_OUT><<<n / 512 + 1, 512>>>(n, in, out);
}

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float,float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal<float,float>(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar<float,float>(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double,double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal<double,double>(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar<double,double>(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<float16,float>(const int n, const float a, const float b,
                                   float16* r) {
  thrust::device_vector<float> rf(n);
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), thrust::raw_pointer_cast(rf.data()), n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal<float,float>(n, range, thrust::raw_pointer_cast(rf.data()));
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar<float,float>(n, a, thrust::raw_pointer_cast(rf.data()));
  }
  caffe_gpu_convert<float,float16>(n, thrust::raw_pointer_cast(rf.data()), r);
}

template <>
void caffe_gpu_rng_uniform<float16,float16>(const int n, const float16 a, const float16 b,
					    float16* r) {
  caffe_gpu_rng_uniform<float16,float>(n, a, b, r);	 
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float16* r) {
  // TODO: call fp16-based version of hiprandGenerateNormal when it becomes available.
  thrust::device_vector<float> rf(n);
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), thrust::raw_pointer_cast(rf.data()), n, mu, sigma));
  caffe_gpu_convert<float,float16>(n, thrust::raw_pointer_cast(rf.data()), r);
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float16 mu, const float16 sigma,
                            float16* r) {
  // TODO: call fp16-based version of hiprandGenerateNormal when it becomes available.
  thrust::device_vector<float> rf(n);
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), thrust::raw_pointer_cast(rf.data()), n, mu, sigma));
  caffe_gpu_convert<float,float16>(n, thrust::raw_pointer_cast(rf.data()), r);
}

}  // namespace caffe
